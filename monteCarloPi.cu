#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include <iostream>
using namespace std;

#define PI 3.1415926535897931f

// Variables needed in pi().
int numThreads; int numBlocks;
int threadNumberOfSamples;

// Hits within the unit circle.
int* m_hits;
// Random number generator on device. 
hiprandState* d_state;

int howMany = 30e6;

// Error checking.
inline void CHECK_ERR( hipError_t err ) {
    if( err != hipSuccess ) {
        const char* errStr = hipGetErrorString( err );
        printf( "%s (%d): %s\n", __FILE__, __LINE__, errStr );
        assert( 0 );
    }
}

// Sequences generated with the same seed and different sequence 
// numbers will not have statistically correlated values.
__global__ void initRandomNumberGenerator( hiprandState* state ){
    int index = threadIdx.x + blockIdx.x * blockDim.x;       
    hiprand_init( 1234, index, 12, &state[index] );
}

void pi_init() {
  int device; int nProc;
  hipGetDevice(&device);
  hipDeviceGetAttribute( &nProc, hipDeviceAttributeMultiprocessorCount, device );
 
  // Use T4 architecture optimized values.
  numThreads = 32;
  numBlocks = nProc * 10; 

  CHECK_ERR( hipMallocManaged( &m_hits, sizeof(int) ) );
  hipMemset( m_hits, 0, sizeof(int) );

  // Use one random number generator pr. thread.
  CHECK_ERR( hipMalloc( &d_state, sizeof(hiprandState) * numThreads * numBlocks ) );

  initRandomNumberGenerator<<< numBlocks, numThreads >>>( d_state );
  CHECK_ERR( hipGetLastError() );
  
  threadNumberOfSamples = ceil( double(howMany) / ( numThreads * numBlocks ) );
}

// Generates two floats on [0,1] and tests (x,y) within the unit circle.
__device__ inline void generateRandomAndUpdate( int& counter, hiprandState& state ) {
  float x = hiprand_uniform( &state );
  float y = hiprand_uniform( &state );

  if( x * x + y * y < 1.f ) 
    ++counter;
}

// The calculation of Pi using Monte Carlo doesn't have correlation between threads.
__global__ void monteCarloPi( int* count, hiprandState* state, const int samples, const int total ) {
  extern __shared__ int smem[];

  auto index = threadIdx.x + blockIdx.x * blockDim.x;
  int globalIndex = index * samples;

  // The number of unit circle hits for this thread.
  int localCounter = 0;
  // Random number generator.
  auto localState = state[index];

  // Each thread takes a portion of samples.
  for( auto i = 0; i < samples && globalIndex + i < total; ++i )
    generateRandomAndUpdate( localCounter, localState );

  // Prepare to sum threads in this block.
  smem[threadIdx.x] = localCounter; // 0 bank conflicts

  __syncthreads();

  // 1 thread handles the block contribution.
  if( threadIdx.x == 0 ){
    localCounter = thrust::reduce( thrust::seq, smem, smem + blockDim.x );
    atomicAdd( count, localCounter );
  }   
	
  state[index] = localState; 
}

// Tesla T4 speedup: a factor of ~3000 over 1 CPU for comparable accuracy.
double pi() {
    *m_hits = 0;
    monteCarloPi<<< numBlocks, numThreads,
                    sizeof(int) * numThreads >>>( m_hits, d_state, threadNumberOfSamples, howMany );
  
    CHECK_ERR( hipGetLastError() );
    CHECK_ERR( hipStreamSynchronize(0) );

    double myPi = 4.0 * (*m_hits) / howMany;
    return myPi;
}

void pi_reset() {
    CHECK_ERR( hipFree( d_state ) );
    CHECK_ERR( hipFree( m_hits ) );
    d_state = nullptr;
    m_hits = nullptr;
}

int main( int argc, char* argv[] ) {
    pi_init();
    cout << fabs( PI - pi() ) << endl;
    pi_reset();

    return 0;
}